#include<iostream>
#include<hip/hip_runtime.h>

#include "cutlass/cutlass.h"
#include "cutlass/layout/matrix.h"
#include "cutlass/gemm/device/gemm_batched.h"
#include <cutlass/util/host_tensor.h>

#include<vector>

void cutlass_strided_bathed_sgemm(
    int m, int n, int k,
    float alpha, float const *A, int lda, long long int batch_stride_A,
    float const *B, int ldb, long long int batch_stride_B,
    float *C, int ldc, long long int batch_stride_C,
    float beta, int batch_count
){
    using Gemm2 = cutlass::gemm::device::GemmBatched<float, cutlass::layout::ColumnMajor,
                                                    float, cutlass::layout::RowMajor,
                                                    float, cutlass::layout::RowMajor,
                                                    float,
                                                    cutlass::arch::OpClassSimt,
                                                    cutlass::arch::Sm50,
                                                    cutlass::gemm::GemmShape<64, 256, 8>,
                                                    cutlass::gemm::GemmShape<32, 64, 8>,
                                                    cutlass::gemm::GemmShape<1, 1, 1>,
                                                    cutlass::epilogue::thread::LinearCombination<float, 1, float, float>,
                                                    cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle,
                                                    2,
                                                    1,
                                                    1,
                                                    true,
                                                    cutlass::arch::OpMultiplyAdd
                                                    >;
    Gemm2 gemm2_op;

    float time;

    Gemm2::Arguments arguments{
        {m, n, k},
        {A, lda}, batch_stride_A,
        {B, ldb}, batch_stride_B,
        {C, ldc}, batch_stride_C,
        {C, ldc}, batch_stride_C,
        {alpha, beta},
        batch_count,
        1
    };

    size_t workspace_size = gemm2_op.get_workspace_size(arguments);
    cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);
    cutlass::Status status = gemm2_op.initialize(arguments, workspace.get());

    for(int i = 0; i < 0; i++){
        gemm2_op();
    }

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);


    hipDeviceSynchronize();
    hipEventRecord(start);
    for(int i = 0; i < 100; i++){
        gemm2_op();   
    }
    hipEventRecord(end);
    hipEventSynchronize(end);

    hipEventElapsedTime(&time, start, end);

     std::cout << time / 100 << std::endl;
}

int main(int argc, char *argv[]){
    int tmp_m = 64;
    int tmp_n = 64;
    int tmp_k = 64;
    int tmp_b = 1;

    //define dimension
    int const m = tmp_m;
    int const n = tmp_n;
    int const k = tmp_k;
    int const batch_count = tmp_b;

    //NT -> T
    int const lda = k;
    int const ldb = k;
    int const ldc = n;

    int const count_A = batch_count * lda * m;
    int const count_B = batch_count * ldb * n;
    int const count_C = batch_count * ldc * m;

    long long int batch_stride_A = static_cast<long long int>(lda) * static_cast<long long int>(m);
    long long int batch_stride_B = static_cast<long long int>(ldb) * static_cast<long long int>(n);
    long long int batch_stride_C = static_cast<long long int>(ldc) * static_cast<long long int>(m);

    float alpha = 1.0f;
    float beta = 0.0f;

    hipError_t result = hipSuccess;

    std::vector<float> host_A(count_A);
    std::vector<float> host_B(count_B);
    std::vector<float> host_C(count_C);

    srand(time(NULL));

    for(int i = 0; i < host_A.size(); i++) host_A[i] = rand() % 100;
    for(int i = 0; i < host_B.size(); i++) host_B[i] = rand() % 100;

    float *A;
    float *B;
    float *C;

    result = hipMalloc(&A, count_A * sizeof(float));
    result = hipMalloc(&B, count_B * sizeof(float));
    result = hipMalloc(&C, count_C * sizeof(float));

    result = hipMemcpy(A, host_A.data(), count_A * sizeof(float), hipMemcpyHostToDevice);
    result = hipMemcpy(B, host_B.data(), count_B * sizeof(float), hipMemcpyHostToDevice);
    result = hipMemcpy(C, host_C.data(), count_C * sizeof(float), hipMemcpyHostToDevice);

    if(result != hipSuccess) std::cout << "not success" << std::endl;

    cutlass_strided_bathed_sgemm(m, n, k, alpha, A, lda, batch_stride_A, B,
                                           ldb, batch_stride_B, C, ldc, batch_stride_C,
                                           beta, batch_count);

    // result = hipMemcpy(host_C.data(), C, count_C * sizeof(float), hipMemcpyDeviceToHost);

    // for(int i = 0; i < host_C.size(); i++) std::cout << host_C[i] << std::endl;

    hipFree(A);
    hipFree(B);
    hipFree(C);

    return 0;
}
