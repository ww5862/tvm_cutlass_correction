
#include<iostream>
#include<hip/hip_runtime.h>

#include <unistd.h>
#include<string>
#include<fstream>      

#include "cutlass/cutlass.h"
#include "cutlass/layout/matrix.h"
#include "cutlass/gemm/device/gemm_batched.h"
#include <cutlass/util/host_tensor.h>


float cutlass_strided_bathed_sgemm_0(
    int m, int n, int k,
    float alpha, float const *A, int lda, long long int batch_stride_A,
    float const *B, int ldb, long long int batch_stride_B,
    float *C, int ldc, long long int batch_stride_C,
    float beta, int batch_count, int split_k, int warmup=0
){
    using Gemm = cutlass::gemm::device::GemmBatched<float, cutlass::layout::RowMajor,
                                                    float, cutlass::layout::RowMajor,
                                                    float, cutlass::layout::RowMajor,
                                                    float,
                                                    cutlass::arch::OpClassSimt,
                                                    cutlass::arch::Sm80,
                                                    cutlass::gemm::GemmShape<64, 64, 8>,
                                                    cutlass::gemm::GemmShape<32, 16, 8>,
                                                    cutlass::gemm::GemmShape<1, 1, 1>,
                                                    cutlass::epilogue::thread::LinearCombination<float, 1, float, float>,
                                                    cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle,
                                                    2,
                                                    1,
                                                    1,
                                                    true,
                                                    cutlass::arch::OpMultiplyAdd
                                                    >;
    
    Gemm gemm_op;
    
    float total_time;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    
    hipDeviceSynchronize();
    hipEventRecord(start);
    
    for(int i = 0; i < 20; i++){
        Gemm::Arguments arguments{
            {m, n, k},
            {A, lda}, batch_stride_A,
            {B, ldb}, batch_stride_B,
            {C, ldc}, batch_stride_C,
            {C, ldc}, batch_stride_C,
            {alpha, beta},
            batch_count,
            2
        };
        
        size_t workspace_size = gemm_op.get_workspace_size(arguments);
        cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

        gemm_op.initialize(arguments, workspace.get());
        cutlass::Status status = gemm_op();
        //workspace.release();
        
        if(status != cutlass::Status::kSuccess){
            hipEventDestroy(start);
            hipEventDestroy(end);
            
            std::fstream dataFile2;
            std::string fileName2 = "/home/local_guest/tvm/python/tvm/contrib/cutlass/rlt_cutlass_TT_Tailor/" + std::to_string(batch_count) + "_" +
                                    std::to_string(m) + "_" + std::to_string(n) + "_" + std::to_string(k) + ".json";
            if(warmup == 0){
                std::string json2 = "{\"dim\": [[64, 64, 8], [32, 16, 8], [2], [1]], \"split_k\": " + std::to_string(split_k)
                                    + " ,\"time\": " + std::to_string(-1) + "}";
                
                dataFile2.open(fileName2, std::ios::app);
                
                dataFile2 << json2 << std::endl;
            }

            return -1;
        }
    }
    
    hipEventRecord(end);
    hipEventSynchronize(end);
    
    hipEventElapsedTime(&total_time, start, end);

    hipEventDestroy(start);
    hipEventDestroy(end);

    hipDeviceSynchronize();
    
    std::fstream dataFile;
    
    std::string fileName = "/home/local_guest/tvm/python/tvm/contrib/cutlass/rlt_cutlass_TT_Tailor/" + std::to_string(batch_count) + "_" +
                            std::to_string(m) + "_" + std::to_string(n) + "_" + std::to_string(k) + ".json";
    if(warmup == 0){
        std::string json = "{\"dim\": [[64, 64, 8], [32, 16, 8], [2], [1]], \"split_k\": " + std::to_string(split_k)
                            + " ,\"time\": " + std::to_string(total_time/20) + "}";
        
        dataFile.open(fileName, std::ios::app);
        
        dataFile << json << std::endl;
    }
    
    return total_time / 20;
    
}
        


float cutlass_strided_bathed_sgemm_1(
    int m, int n, int k,
    float alpha, float const *A, int lda, long long int batch_stride_A,
    float const *B, int ldb, long long int batch_stride_B,
    float *C, int ldc, long long int batch_stride_C,
    float beta, int batch_count, int split_k, int warmup=0
){
    using Gemm = cutlass::gemm::device::GemmBatched<float, cutlass::layout::RowMajor,
                                                    float, cutlass::layout::RowMajor,
                                                    float, cutlass::layout::RowMajor,
                                                    float,
                                                    cutlass::arch::OpClassSimt,
                                                    cutlass::arch::Sm80,
                                                    cutlass::gemm::GemmShape<128, 64, 8>,
                                                    cutlass::gemm::GemmShape<128, 16, 8>,
                                                    cutlass::gemm::GemmShape<1, 1, 1>,
                                                    cutlass::epilogue::thread::LinearCombination<float, 1, float, float>,
                                                    cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle,
                                                    2,
                                                    1,
                                                    1,
                                                    true,
                                                    cutlass::arch::OpMultiplyAdd
                                                    >;
    
    Gemm gemm_op;
    
    float total_time;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    
    hipDeviceSynchronize();
    hipEventRecord(start);
    
    for(int i = 0; i < 20; i++){
        Gemm::Arguments arguments{
            {m, n, k},
            {A, lda}, batch_stride_A,
            {B, ldb}, batch_stride_B,
            {C, ldc}, batch_stride_C,
            {C, ldc}, batch_stride_C,
            {alpha, beta},
            batch_count,
            4
        };
        
        size_t workspace_size = gemm_op.get_workspace_size(arguments);
        cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

        gemm_op.initialize(arguments, workspace.get());
        cutlass::Status status = gemm_op();
        //workspace.release();
        
        if(status != cutlass::Status::kSuccess){
            hipEventDestroy(start);
            hipEventDestroy(end);
            
            std::fstream dataFile2;
            std::string fileName2 = "/home/local_guest/tvm/python/tvm/contrib/cutlass/rlt_cutlass_TT_Tailor/" + std::to_string(batch_count) + "_" +
                                    std::to_string(m) + "_" + std::to_string(n) + "_" + std::to_string(k) + ".json";
            if(warmup == 0){
                std::string json2 = "{\"dim\": [[128, 64, 8], [128, 16, 8], [2], [1]], \"split_k\": " + std::to_string(split_k)
                                    + " ,\"time\": " + std::to_string(-1) + "}";
                
                dataFile2.open(fileName2, std::ios::app);
                
                dataFile2 << json2 << std::endl;
            }

            return -1;
        }
    }
    
    hipEventRecord(end);
    hipEventSynchronize(end);
    
    hipEventElapsedTime(&total_time, start, end);

    hipEventDestroy(start);
    hipEventDestroy(end);

    hipDeviceSynchronize();
    
    std::fstream dataFile;
    
    std::string fileName = "/home/local_guest/tvm/python/tvm/contrib/cutlass/rlt_cutlass_TT_Tailor/" + std::to_string(batch_count) + "_" +
                            std::to_string(m) + "_" + std::to_string(n) + "_" + std::to_string(k) + ".json";
    if(warmup == 0){
        std::string json = "{\"dim\": [[128, 64, 8], [128, 16, 8], [2], [1]], \"split_k\": " + std::to_string(split_k)
                            + " ,\"time\": " + std::to_string(total_time/20) + "}";
        
        dataFile.open(fileName, std::ios::app);
        
        dataFile << json << std::endl;
    }
    
    return total_time / 20;
    
}
        


float cutlass_strided_bathed_sgemm_2(
    int m, int n, int k,
    float alpha, float const *A, int lda, long long int batch_stride_A,
    float const *B, int ldb, long long int batch_stride_B,
    float *C, int ldc, long long int batch_stride_C,
    float beta, int batch_count, int split_k, int warmup=0
){
    using Gemm = cutlass::gemm::device::GemmBatched<float, cutlass::layout::RowMajor,
                                                    float, cutlass::layout::RowMajor,
                                                    float, cutlass::layout::RowMajor,
                                                    float,
                                                    cutlass::arch::OpClassSimt,
                                                    cutlass::arch::Sm80,
                                                    cutlass::gemm::GemmShape<64, 64, 8>,
                                                    cutlass::gemm::GemmShape<16, 32, 8>,
                                                    cutlass::gemm::GemmShape<1, 1, 1>,
                                                    cutlass::epilogue::thread::LinearCombination<float, 1, float, float>,
                                                    cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle,
                                                    2,
                                                    1,
                                                    1,
                                                    true,
                                                    cutlass::arch::OpMultiplyAdd
                                                    >;
    
    Gemm gemm_op;
    
    float total_time;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    
    hipDeviceSynchronize();
    hipEventRecord(start);
    
    for(int i = 0; i < 20; i++){
        Gemm::Arguments arguments{
            {m, n, k},
            {A, lda}, batch_stride_A,
            {B, ldb}, batch_stride_B,
            {C, ldc}, batch_stride_C,
            {C, ldc}, batch_stride_C,
            {alpha, beta},
            batch_count,
            2
        };
        
        size_t workspace_size = gemm_op.get_workspace_size(arguments);
        cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

        gemm_op.initialize(arguments, workspace.get());
        cutlass::Status status = gemm_op();
        //workspace.release();
        
        if(status != cutlass::Status::kSuccess){
            hipEventDestroy(start);
            hipEventDestroy(end);
            
            std::fstream dataFile2;
            std::string fileName2 = "/home/local_guest/tvm/python/tvm/contrib/cutlass/rlt_cutlass_TT_Tailor/" + std::to_string(batch_count) + "_" +
                                    std::to_string(m) + "_" + std::to_string(n) + "_" + std::to_string(k) + ".json";
            if(warmup == 0){
                std::string json2 = "{\"dim\": [[64, 64, 8], [16, 32, 8], [2], [1]], \"split_k\": " + std::to_string(split_k)
                                    + " ,\"time\": " + std::to_string(-1) + "}";
                
                dataFile2.open(fileName2, std::ios::app);
                
                dataFile2 << json2 << std::endl;
            }

            return -1;
        }
    }
    
    hipEventRecord(end);
    hipEventSynchronize(end);
    
    hipEventElapsedTime(&total_time, start, end);

    hipEventDestroy(start);
    hipEventDestroy(end);

    hipDeviceSynchronize();
    
    std::fstream dataFile;
    
    std::string fileName = "/home/local_guest/tvm/python/tvm/contrib/cutlass/rlt_cutlass_TT_Tailor/" + std::to_string(batch_count) + "_" +
                            std::to_string(m) + "_" + std::to_string(n) + "_" + std::to_string(k) + ".json";
    if(warmup == 0){
        std::string json = "{\"dim\": [[64, 64, 8], [16, 32, 8], [2], [1]], \"split_k\": " + std::to_string(split_k)
                            + " ,\"time\": " + std::to_string(total_time/20) + "}";
        
        dataFile.open(fileName, std::ios::app);
        
        dataFile << json << std::endl;
    }
    
    return total_time / 20;
    
}
        


float cutlass_strided_bathed_sgemm_3(
    int m, int n, int k,
    float alpha, float const *A, int lda, long long int batch_stride_A,
    float const *B, int ldb, long long int batch_stride_B,
    float *C, int ldc, long long int batch_stride_C,
    float beta, int batch_count, int split_k, int warmup=0
){
    using Gemm = cutlass::gemm::device::GemmBatched<float, cutlass::layout::RowMajor,
                                                    float, cutlass::layout::RowMajor,
                                                    float, cutlass::layout::RowMajor,
                                                    float,
                                                    cutlass::arch::OpClassSimt,
                                                    cutlass::arch::Sm80,
                                                    cutlass::gemm::GemmShape<64, 64, 8>,
                                                    cutlass::gemm::GemmShape<32, 16, 8>,
                                                    cutlass::gemm::GemmShape<1, 1, 1>,
                                                    cutlass::epilogue::thread::LinearCombination<float, 1, float, float>,
                                                    cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle,
                                                    2,
                                                    1,
                                                    1,
                                                    true,
                                                    cutlass::arch::OpMultiplyAdd
                                                    >;
    
    Gemm gemm_op;
    
    float total_time;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    
    hipDeviceSynchronize();
    hipEventRecord(start);
    
    for(int i = 0; i < 20; i++){
        Gemm::Arguments arguments{
            {m, n, k},
            {A, lda}, batch_stride_A,
            {B, ldb}, batch_stride_B,
            {C, ldc}, batch_stride_C,
            {C, ldc}, batch_stride_C,
            {alpha, beta},
            batch_count,
            1
        };
        
        size_t workspace_size = gemm_op.get_workspace_size(arguments);
        cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

        gemm_op.initialize(arguments, workspace.get());
        cutlass::Status status = gemm_op();
        //workspace.release();
        
        if(status != cutlass::Status::kSuccess){
            hipEventDestroy(start);
            hipEventDestroy(end);
            
            std::fstream dataFile2;
            std::string fileName2 = "/home/local_guest/tvm/python/tvm/contrib/cutlass/rlt_cutlass_TT_Tailor/" + std::to_string(batch_count) + "_" +
                                    std::to_string(m) + "_" + std::to_string(n) + "_" + std::to_string(k) + ".json";
            if(warmup == 0){
                std::string json2 = "{\"dim\": [[64, 64, 8], [32, 16, 8], [2], [1]], \"split_k\": " + std::to_string(split_k)
                                    + " ,\"time\": " + std::to_string(-1) + "}";
                
                dataFile2.open(fileName2, std::ios::app);
                
                dataFile2 << json2 << std::endl;
            }

            return -1;
        }
    }
    
    hipEventRecord(end);
    hipEventSynchronize(end);
    
    hipEventElapsedTime(&total_time, start, end);

    hipEventDestroy(start);
    hipEventDestroy(end);

    hipDeviceSynchronize();
    
    std::fstream dataFile;
    
    std::string fileName = "/home/local_guest/tvm/python/tvm/contrib/cutlass/rlt_cutlass_TT_Tailor/" + std::to_string(batch_count) + "_" +
                            std::to_string(m) + "_" + std::to_string(n) + "_" + std::to_string(k) + ".json";
    if(warmup == 0){
        std::string json = "{\"dim\": [[64, 64, 8], [32, 16, 8], [2], [1]], \"split_k\": " + std::to_string(split_k)
                            + " ,\"time\": " + std::to_string(total_time/20) + "}";
        
        dataFile.open(fileName, std::ios::app);
        
        dataFile << json << std::endl;
    }
    
    return total_time / 20;
    
}
        


float cutlass_strided_bathed_sgemm_4(
    int m, int n, int k,
    float alpha, float const *A, int lda, long long int batch_stride_A,
    float const *B, int ldb, long long int batch_stride_B,
    float *C, int ldc, long long int batch_stride_C,
    float beta, int batch_count, int split_k, int warmup=0
){
    using Gemm = cutlass::gemm::device::GemmBatched<float, cutlass::layout::RowMajor,
                                                    float, cutlass::layout::RowMajor,
                                                    float, cutlass::layout::RowMajor,
                                                    float,
                                                    cutlass::arch::OpClassSimt,
                                                    cutlass::arch::Sm80,
                                                    cutlass::gemm::GemmShape<64, 64, 8>,
                                                    cutlass::gemm::GemmShape<16, 32, 8>,
                                                    cutlass::gemm::GemmShape<1, 1, 1>,
                                                    cutlass::epilogue::thread::LinearCombination<float, 1, float, float>,
                                                    cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle,
                                                    2,
                                                    1,
                                                    1,
                                                    true,
                                                    cutlass::arch::OpMultiplyAdd
                                                    >;
    
    Gemm gemm_op;
    
    float total_time;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    
    hipDeviceSynchronize();
    hipEventRecord(start);
    
    for(int i = 0; i < 20; i++){
        Gemm::Arguments arguments{
            {m, n, k},
            {A, lda}, batch_stride_A,
            {B, ldb}, batch_stride_B,
            {C, ldc}, batch_stride_C,
            {C, ldc}, batch_stride_C,
            {alpha, beta},
            batch_count,
            1
        };
        
        size_t workspace_size = gemm_op.get_workspace_size(arguments);
        cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

        gemm_op.initialize(arguments, workspace.get());
        cutlass::Status status = gemm_op();
        //workspace.release();
        
        if(status != cutlass::Status::kSuccess){
            hipEventDestroy(start);
            hipEventDestroy(end);
            
            std::fstream dataFile2;
            std::string fileName2 = "/home/local_guest/tvm/python/tvm/contrib/cutlass/rlt_cutlass_TT_Tailor/" + std::to_string(batch_count) + "_" +
                                    std::to_string(m) + "_" + std::to_string(n) + "_" + std::to_string(k) + ".json";
            if(warmup == 0){
                std::string json2 = "{\"dim\": [[64, 64, 8], [16, 32, 8], [2], [1]], \"split_k\": " + std::to_string(split_k)
                                    + " ,\"time\": " + std::to_string(-1) + "}";
                
                dataFile2.open(fileName2, std::ios::app);
                
                dataFile2 << json2 << std::endl;
            }

            return -1;
        }
    }
    
    hipEventRecord(end);
    hipEventSynchronize(end);
    
    hipEventElapsedTime(&total_time, start, end);

    hipEventDestroy(start);
    hipEventDestroy(end);

    hipDeviceSynchronize();
    
    std::fstream dataFile;
    
    std::string fileName = "/home/local_guest/tvm/python/tvm/contrib/cutlass/rlt_cutlass_TT_Tailor/" + std::to_string(batch_count) + "_" +
                            std::to_string(m) + "_" + std::to_string(n) + "_" + std::to_string(k) + ".json";
    if(warmup == 0){
        std::string json = "{\"dim\": [[64, 64, 8], [16, 32, 8], [2], [1]], \"split_k\": " + std::to_string(split_k)
                            + " ,\"time\": " + std::to_string(total_time/20) + "}";
        
        dataFile.open(fileName, std::ios::app);
        
        dataFile << json << std::endl;
    }
    
    return total_time / 20;
    
}
        


float cutlass_strided_bathed_sgemm_5(
    int m, int n, int k,
    float alpha, float const *A, int lda, long long int batch_stride_A,
    float const *B, int ldb, long long int batch_stride_B,
    float *C, int ldc, long long int batch_stride_C,
    float beta, int batch_count, int split_k, int warmup=0
){
    using Gemm = cutlass::gemm::device::GemmBatched<float, cutlass::layout::RowMajor,
                                                    float, cutlass::layout::RowMajor,
                                                    float, cutlass::layout::RowMajor,
                                                    float,
                                                    cutlass::arch::OpClassSimt,
                                                    cutlass::arch::Sm80,
                                                    cutlass::gemm::GemmShape<64, 64, 8>,
                                                    cutlass::gemm::GemmShape<32, 16, 8>,
                                                    cutlass::gemm::GemmShape<1, 1, 1>,
                                                    cutlass::epilogue::thread::LinearCombination<float, 1, float, float>,
                                                    cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle,
                                                    2,
                                                    1,
                                                    1,
                                                    true,
                                                    cutlass::arch::OpMultiplyAdd
                                                    >;
    
    Gemm gemm_op;
    
    float total_time;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    
    hipDeviceSynchronize();
    hipEventRecord(start);
    
    for(int i = 0; i < 20; i++){
        Gemm::Arguments arguments{
            {m, n, k},
            {A, lda}, batch_stride_A,
            {B, ldb}, batch_stride_B,
            {C, ldc}, batch_stride_C,
            {C, ldc}, batch_stride_C,
            {alpha, beta},
            batch_count,
            2
        };
        
        size_t workspace_size = gemm_op.get_workspace_size(arguments);
        cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

        gemm_op.initialize(arguments, workspace.get());
        cutlass::Status status = gemm_op();
        //workspace.release();
        
        if(status != cutlass::Status::kSuccess){
            hipEventDestroy(start);
            hipEventDestroy(end);
            
            std::fstream dataFile2;
            std::string fileName2 = "/home/local_guest/tvm/python/tvm/contrib/cutlass/rlt_cutlass_TT_Tailor/" + std::to_string(batch_count) + "_" +
                                    std::to_string(m) + "_" + std::to_string(n) + "_" + std::to_string(k) + ".json";
            if(warmup == 0){
                std::string json2 = "{\"dim\": [[64, 64, 8], [32, 16, 8], [2], [1]], \"split_k\": " + std::to_string(split_k)
                                    + " ,\"time\": " + std::to_string(-1) + "}";
                
                dataFile2.open(fileName2, std::ios::app);
                
                dataFile2 << json2 << std::endl;
            }

            return -1;
        }
    }
    
    hipEventRecord(end);
    hipEventSynchronize(end);
    
    hipEventElapsedTime(&total_time, start, end);

    hipEventDestroy(start);
    hipEventDestroy(end);

    hipDeviceSynchronize();
    
    std::fstream dataFile;
    
    std::string fileName = "/home/local_guest/tvm/python/tvm/contrib/cutlass/rlt_cutlass_TT_Tailor/" + std::to_string(batch_count) + "_" +
                            std::to_string(m) + "_" + std::to_string(n) + "_" + std::to_string(k) + ".json";
    if(warmup == 0){
        std::string json = "{\"dim\": [[64, 64, 8], [32, 16, 8], [2], [1]], \"split_k\": " + std::to_string(split_k)
                            + " ,\"time\": " + std::to_string(total_time/20) + "}";
        
        dataFile.open(fileName, std::ios::app);
        
        dataFile << json << std::endl;
    }
    
    return total_time / 20;
    
}
        


float cutlass_strided_bathed_sgemm_6(
    int m, int n, int k,
    float alpha, float const *A, int lda, long long int batch_stride_A,
    float const *B, int ldb, long long int batch_stride_B,
    float *C, int ldc, long long int batch_stride_C,
    float beta, int batch_count, int split_k, int warmup=0
){
    using Gemm = cutlass::gemm::device::GemmBatched<float, cutlass::layout::RowMajor,
                                                    float, cutlass::layout::RowMajor,
                                                    float, cutlass::layout::RowMajor,
                                                    float,
                                                    cutlass::arch::OpClassSimt,
                                                    cutlass::arch::Sm80,
                                                    cutlass::gemm::GemmShape<128, 64, 8>,
                                                    cutlass::gemm::GemmShape<128, 16, 8>,
                                                    cutlass::gemm::GemmShape<1, 1, 1>,
                                                    cutlass::epilogue::thread::LinearCombination<float, 1, float, float>,
                                                    cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle,
                                                    2,
                                                    1,
                                                    1,
                                                    true,
                                                    cutlass::arch::OpMultiplyAdd
                                                    >;
    
    Gemm gemm_op;
    
    float total_time;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    
    hipDeviceSynchronize();
    hipEventRecord(start);
    
    for(int i = 0; i < 20; i++){
        Gemm::Arguments arguments{
            {m, n, k},
            {A, lda}, batch_stride_A,
            {B, ldb}, batch_stride_B,
            {C, ldc}, batch_stride_C,
            {C, ldc}, batch_stride_C,
            {alpha, beta},
            batch_count,
            4
        };
        
        size_t workspace_size = gemm_op.get_workspace_size(arguments);
        cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

        gemm_op.initialize(arguments, workspace.get());
        cutlass::Status status = gemm_op();
        //workspace.release();
        
        if(status != cutlass::Status::kSuccess){
            hipEventDestroy(start);
            hipEventDestroy(end);
            
            std::fstream dataFile2;
            std::string fileName2 = "/home/local_guest/tvm/python/tvm/contrib/cutlass/rlt_cutlass_TT_Tailor/" + std::to_string(batch_count) + "_" +
                                    std::to_string(m) + "_" + std::to_string(n) + "_" + std::to_string(k) + ".json";
            if(warmup == 0){
                std::string json2 = "{\"dim\": [[128, 64, 8], [128, 16, 8], [2], [1]], \"split_k\": " + std::to_string(split_k)
                                    + " ,\"time\": " + std::to_string(-1) + "}";
                
                dataFile2.open(fileName2, std::ios::app);
                
                dataFile2 << json2 << std::endl;
            }

            return -1;
        }
    }
    
    hipEventRecord(end);
    hipEventSynchronize(end);
    
    hipEventElapsedTime(&total_time, start, end);

    hipEventDestroy(start);
    hipEventDestroy(end);

    hipDeviceSynchronize();
    
    std::fstream dataFile;
    
    std::string fileName = "/home/local_guest/tvm/python/tvm/contrib/cutlass/rlt_cutlass_TT_Tailor/" + std::to_string(batch_count) + "_" +
                            std::to_string(m) + "_" + std::to_string(n) + "_" + std::to_string(k) + ".json";
    if(warmup == 0){
        std::string json = "{\"dim\": [[128, 64, 8], [128, 16, 8], [2], [1]], \"split_k\": " + std::to_string(split_k)
                            + " ,\"time\": " + std::to_string(total_time/20) + "}";
        
        dataFile.open(fileName, std::ios::app);
        
        dataFile << json << std::endl;
    }
    
    return total_time / 20;
    
}
        


float cutlass_strided_bathed_sgemm_7(
    int m, int n, int k,
    float alpha, float const *A, int lda, long long int batch_stride_A,
    float const *B, int ldb, long long int batch_stride_B,
    float *C, int ldc, long long int batch_stride_C,
    float beta, int batch_count, int split_k, int warmup=0
){
    using Gemm = cutlass::gemm::device::GemmBatched<float, cutlass::layout::RowMajor,
                                                    float, cutlass::layout::RowMajor,
                                                    float, cutlass::layout::RowMajor,
                                                    float,
                                                    cutlass::arch::OpClassSimt,
                                                    cutlass::arch::Sm80,
                                                    cutlass::gemm::GemmShape<64, 64, 8>,
                                                    cutlass::gemm::GemmShape<16, 32, 8>,
                                                    cutlass::gemm::GemmShape<1, 1, 1>,
                                                    cutlass::epilogue::thread::LinearCombination<float, 1, float, float>,
                                                    cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle,
                                                    2,
                                                    1,
                                                    1,
                                                    true,
                                                    cutlass::arch::OpMultiplyAdd
                                                    >;
    
    Gemm gemm_op;
    
    float total_time;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    
    hipDeviceSynchronize();
    hipEventRecord(start);
    
    for(int i = 0; i < 20; i++){
        Gemm::Arguments arguments{
            {m, n, k},
            {A, lda}, batch_stride_A,
            {B, ldb}, batch_stride_B,
            {C, ldc}, batch_stride_C,
            {C, ldc}, batch_stride_C,
            {alpha, beta},
            batch_count,
            2
        };
        
        size_t workspace_size = gemm_op.get_workspace_size(arguments);
        cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

        gemm_op.initialize(arguments, workspace.get());
        cutlass::Status status = gemm_op();
        //workspace.release();
        
        if(status != cutlass::Status::kSuccess){
            hipEventDestroy(start);
            hipEventDestroy(end);
            
            std::fstream dataFile2;
            std::string fileName2 = "/home/local_guest/tvm/python/tvm/contrib/cutlass/rlt_cutlass_TT_Tailor/" + std::to_string(batch_count) + "_" +
                                    std::to_string(m) + "_" + std::to_string(n) + "_" + std::to_string(k) + ".json";
            if(warmup == 0){
                std::string json2 = "{\"dim\": [[64, 64, 8], [16, 32, 8], [2], [1]], \"split_k\": " + std::to_string(split_k)
                                    + " ,\"time\": " + std::to_string(-1) + "}";
                
                dataFile2.open(fileName2, std::ios::app);
                
                dataFile2 << json2 << std::endl;
            }

            return -1;
        }
    }
    
    hipEventRecord(end);
    hipEventSynchronize(end);
    
    hipEventElapsedTime(&total_time, start, end);

    hipEventDestroy(start);
    hipEventDestroy(end);

    hipDeviceSynchronize();
    
    std::fstream dataFile;
    
    std::string fileName = "/home/local_guest/tvm/python/tvm/contrib/cutlass/rlt_cutlass_TT_Tailor/" + std::to_string(batch_count) + "_" +
                            std::to_string(m) + "_" + std::to_string(n) + "_" + std::to_string(k) + ".json";
    if(warmup == 0){
        std::string json = "{\"dim\": [[64, 64, 8], [16, 32, 8], [2], [1]], \"split_k\": " + std::to_string(split_k)
                            + " ,\"time\": " + std::to_string(total_time/20) + "}";
        
        dataFile.open(fileName, std::ios::app);
        
        dataFile << json << std::endl;
    }
    
    return total_time / 20;
    
}
        


float cutlass_strided_bathed_sgemm_8(
    int m, int n, int k,
    float alpha, float const *A, int lda, long long int batch_stride_A,
    float const *B, int ldb, long long int batch_stride_B,
    float *C, int ldc, long long int batch_stride_C,
    float beta, int batch_count, int split_k, int warmup=0
){
    using Gemm = cutlass::gemm::device::GemmBatched<float, cutlass::layout::RowMajor,
                                                    float, cutlass::layout::RowMajor,
                                                    float, cutlass::layout::RowMajor,
                                                    float,
                                                    cutlass::arch::OpClassSimt,
                                                    cutlass::arch::Sm80,
                                                    cutlass::gemm::GemmShape<64, 64, 8>,
                                                    cutlass::gemm::GemmShape<32, 16, 8>,
                                                    cutlass::gemm::GemmShape<1, 1, 1>,
                                                    cutlass::epilogue::thread::LinearCombination<float, 1, float, float>,
                                                    cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle,
                                                    2,
                                                    1,
                                                    1,
                                                    true,
                                                    cutlass::arch::OpMultiplyAdd
                                                    >;
    
    Gemm gemm_op;
    
    float total_time;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    
    hipDeviceSynchronize();
    hipEventRecord(start);
    
    for(int i = 0; i < 20; i++){
        Gemm::Arguments arguments{
            {m, n, k},
            {A, lda}, batch_stride_A,
            {B, ldb}, batch_stride_B,
            {C, ldc}, batch_stride_C,
            {C, ldc}, batch_stride_C,
            {alpha, beta},
            batch_count,
            1
        };
        
        size_t workspace_size = gemm_op.get_workspace_size(arguments);
        cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

        gemm_op.initialize(arguments, workspace.get());
        cutlass::Status status = gemm_op();
        //workspace.release();
        
        if(status != cutlass::Status::kSuccess){
            hipEventDestroy(start);
            hipEventDestroy(end);
            
            std::fstream dataFile2;
            std::string fileName2 = "/home/local_guest/tvm/python/tvm/contrib/cutlass/rlt_cutlass_TT_Tailor/" + std::to_string(batch_count) + "_" +
                                    std::to_string(m) + "_" + std::to_string(n) + "_" + std::to_string(k) + ".json";
            if(warmup == 0){
                std::string json2 = "{\"dim\": [[64, 64, 8], [32, 16, 8], [2], [1]], \"split_k\": " + std::to_string(split_k)
                                    + " ,\"time\": " + std::to_string(-1) + "}";
                
                dataFile2.open(fileName2, std::ios::app);
                
                dataFile2 << json2 << std::endl;
            }

            return -1;
        }
    }
    
    hipEventRecord(end);
    hipEventSynchronize(end);
    
    hipEventElapsedTime(&total_time, start, end);

    hipEventDestroy(start);
    hipEventDestroy(end);

    hipDeviceSynchronize();
    
    std::fstream dataFile;
    
    std::string fileName = "/home/local_guest/tvm/python/tvm/contrib/cutlass/rlt_cutlass_TT_Tailor/" + std::to_string(batch_count) + "_" +
                            std::to_string(m) + "_" + std::to_string(n) + "_" + std::to_string(k) + ".json";
    if(warmup == 0){
        std::string json = "{\"dim\": [[64, 64, 8], [32, 16, 8], [2], [1]], \"split_k\": " + std::to_string(split_k)
                            + " ,\"time\": " + std::to_string(total_time/20) + "}";
        
        dataFile.open(fileName, std::ios::app);
        
        dataFile << json << std::endl;
    }
    
    return total_time / 20;
    
}
        


float cutlass_strided_bathed_sgemm_9(
    int m, int n, int k,
    float alpha, float const *A, int lda, long long int batch_stride_A,
    float const *B, int ldb, long long int batch_stride_B,
    float *C, int ldc, long long int batch_stride_C,
    float beta, int batch_count, int split_k, int warmup=0
){
    using Gemm = cutlass::gemm::device::GemmBatched<float, cutlass::layout::RowMajor,
                                                    float, cutlass::layout::RowMajor,
                                                    float, cutlass::layout::RowMajor,
                                                    float,
                                                    cutlass::arch::OpClassSimt,
                                                    cutlass::arch::Sm80,
                                                    cutlass::gemm::GemmShape<64, 64, 8>,
                                                    cutlass::gemm::GemmShape<16, 32, 8>,
                                                    cutlass::gemm::GemmShape<1, 1, 1>,
                                                    cutlass::epilogue::thread::LinearCombination<float, 1, float, float>,
                                                    cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle,
                                                    2,
                                                    1,
                                                    1,
                                                    true,
                                                    cutlass::arch::OpMultiplyAdd
                                                    >;
    
    Gemm gemm_op;
    
    float total_time;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    
    hipDeviceSynchronize();
    hipEventRecord(start);
    
    for(int i = 0; i < 20; i++){
        Gemm::Arguments arguments{
            {m, n, k},
            {A, lda}, batch_stride_A,
            {B, ldb}, batch_stride_B,
            {C, ldc}, batch_stride_C,
            {C, ldc}, batch_stride_C,
            {alpha, beta},
            batch_count,
            1
        };
        
        size_t workspace_size = gemm_op.get_workspace_size(arguments);
        cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

        gemm_op.initialize(arguments, workspace.get());
        cutlass::Status status = gemm_op();
        //workspace.release();
        
        if(status != cutlass::Status::kSuccess){
            hipEventDestroy(start);
            hipEventDestroy(end);
            
            std::fstream dataFile2;
            std::string fileName2 = "/home/local_guest/tvm/python/tvm/contrib/cutlass/rlt_cutlass_TT_Tailor/" + std::to_string(batch_count) + "_" +
                                    std::to_string(m) + "_" + std::to_string(n) + "_" + std::to_string(k) + ".json";
            if(warmup == 0){
                std::string json2 = "{\"dim\": [[64, 64, 8], [16, 32, 8], [2], [1]], \"split_k\": " + std::to_string(split_k)
                                    + " ,\"time\": " + std::to_string(-1) + "}";
                
                dataFile2.open(fileName2, std::ios::app);
                
                dataFile2 << json2 << std::endl;
            }

            return -1;
        }
    }
    
    hipEventRecord(end);
    hipEventSynchronize(end);
    
    hipEventElapsedTime(&total_time, start, end);

    hipEventDestroy(start);
    hipEventDestroy(end);

    hipDeviceSynchronize();
    
    std::fstream dataFile;
    
    std::string fileName = "/home/local_guest/tvm/python/tvm/contrib/cutlass/rlt_cutlass_TT_Tailor/" + std::to_string(batch_count) + "_" +
                            std::to_string(m) + "_" + std::to_string(n) + "_" + std::to_string(k) + ".json";
    if(warmup == 0){
        std::string json = "{\"dim\": [[64, 64, 8], [16, 32, 8], [2], [1]], \"split_k\": " + std::to_string(split_k)
                            + " ,\"time\": " + std::to_string(total_time/20) + "}";
        
        dataFile.open(fileName, std::ios::app);
        
        dataFile << json << std::endl;
    }
    
    return total_time / 20;
    
}
        



int main(int argc, char *argv[]){
    float *rlt = new float[10];

    int M = 64;
    int N = 64;
    int K = 64;
    int Batch = 1;
    int split_k = 1;
    
    int option;
    while((option = getopt(argc, argv, "m:n:k:b:s:")) != -1){
        switch(option){
            case 'm':
                M = std::stoi(optarg);
                break;
            case 'n':
                N = std::stoi(optarg);
                break;
            case 'k':
                K = std::stoi(optarg);
                break;
            case 'b':
                Batch = std::stoi(optarg);
                break;
            case 's':
                split_k = std::stoi(optarg);
            case '?':
                break;
        }
    }
    
    int const lda = K;
    int const ldb = N;
    int const ldc = N;
    
    int const count_A = Batch * M * K;
    int const count_B = Batch * N * K;
    int const count_C = Batch * M * N;
    
    long long int batch_stride_A = static_cast<long long int>(M) * static_cast<long long int>(K);
    long long int batch_stride_B = static_cast<long long int>(K) * static_cast<long long int>(N);
    long long int batch_stride_C = static_cast<long long int>(M) * static_cast<long long int>(N);
    
    float alpha = static_cast<float>(1.0f);
    float beta = static_cast<float>(0.0f);
    
    std::vector<float> host_A(count_A, 1.2f);
    std::vector<float> host_B(count_B, 1.0f);
    std::vector<float> host_C(count_C);
    
    float *A;
    float *B;
    float *C;
    
    hipMalloc(&A, count_A * sizeof(float));
    hipMalloc(&B, count_B * sizeof(float));
    hipMalloc(&C, count_C * sizeof(float));
    
    hipMemcpy(A, host_A.data(), count_A * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B, host_B.data(), count_B * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(C, host_C.data(), count_C * sizeof(float), hipMemcpyHostToDevice);
    
    //warmp up
    for(int i = 0; i < 20; i++){
        cutlass_strided_bathed_sgemm_0(M, N, K, alpha, A, lda, batch_stride_A, B, ldb, batch_stride_B, C, ldc, batch_stride_C, beta, Batch, 1, 1);
    }
    
    rlt[0] = cutlass_strided_bathed_sgemm_0(M, N, K, alpha, A, lda, batch_stride_A, B, ldb, batch_stride_B, C, ldc, batch_stride_C, beta, Batch, split_k);

	rlt[1] = cutlass_strided_bathed_sgemm_1(M, N, K, alpha, A, lda, batch_stride_A, B, ldb, batch_stride_B, C, ldc, batch_stride_C, beta, Batch, split_k);

	rlt[2] = cutlass_strided_bathed_sgemm_2(M, N, K, alpha, A, lda, batch_stride_A, B, ldb, batch_stride_B, C, ldc, batch_stride_C, beta, Batch, split_k);

	rlt[3] = cutlass_strided_bathed_sgemm_3(M, N, K, alpha, A, lda, batch_stride_A, B, ldb, batch_stride_B, C, ldc, batch_stride_C, beta, Batch, split_k);

	rlt[4] = cutlass_strided_bathed_sgemm_4(M, N, K, alpha, A, lda, batch_stride_A, B, ldb, batch_stride_B, C, ldc, batch_stride_C, beta, Batch, split_k);

	rlt[5] = cutlass_strided_bathed_sgemm_5(M, N, K, alpha, A, lda, batch_stride_A, B, ldb, batch_stride_B, C, ldc, batch_stride_C, beta, Batch, split_k);

	rlt[6] = cutlass_strided_bathed_sgemm_6(M, N, K, alpha, A, lda, batch_stride_A, B, ldb, batch_stride_B, C, ldc, batch_stride_C, beta, Batch, split_k);

	rlt[7] = cutlass_strided_bathed_sgemm_7(M, N, K, alpha, A, lda, batch_stride_A, B, ldb, batch_stride_B, C, ldc, batch_stride_C, beta, Batch, split_k);

	rlt[8] = cutlass_strided_bathed_sgemm_8(M, N, K, alpha, A, lda, batch_stride_A, B, ldb, batch_stride_B, C, ldc, batch_stride_C, beta, Batch, split_k);

	rlt[9] = cutlass_strided_bathed_sgemm_9(M, N, K, alpha, A, lda, batch_stride_A, B, ldb, batch_stride_B, C, ldc, batch_stride_C, beta, Batch, split_k);

	
    
    hipFree(A);
    hipFree(B);
    hipFree(C);
    
    return 0;
}
        